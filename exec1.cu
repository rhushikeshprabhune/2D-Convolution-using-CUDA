#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <sstream>
#include <iomanip>

using namespace std;

//input, filter and output considering maximum dimension possible
double a[1000][1000], h[10][10], c[1000][1000];

/**
 * CUDA Kernel Device code
 *
 * Computes the 2D convolution of input b with filter h and output c
 */

 __global__ void
 convolution2D(double *a_1, double *h_1, double *c_1, int rows_a, int columns_a, int rows_h, int columns_h)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    double sum;

    if((idx < (columns_h+columns_a - 1)) && (idy < (rows_h + rows_a - 1)))
    {
        sum = 0;
        for(int k=0; k< rows_h; k++)
        {
            for(int j=0; j< columns_h; j++)
            {
                if ((idy - k) >= 0 && (idx - j) >= 0 && (idy - k) < rows_a && (idx - j) < columns_a) 
                {
                    sum += a_1[((idy - k)*columns_a) + (idx - j)] * h_1[k*columns_h + j];
                }

            }
        }
        c_1[idy*(columns_a + columns_h - 1) + idx] = sum;
        __syncthreads();
    }
}

/**
 * Host main routine
 */
 int main(int argc, char** argv)
 {
     // Error code to check return values for CUDA calls
     hipError_t err = hipSuccess;

     int columns_a, columns_h, rows_a, rows_h, size_a, size_h;
     columns_a = columns_h = 0;
     size_a = size_h = rows_a = rows_h = 0;
     char *ip_file;
     string line;
     double input;
     ip_file = argv[1];
     int i, j, k;

     ifstream file(ip_file);
     if(file.is_open())
     {
         i=0;
         while(getline(file, line) && line != "")
         {
              j=0;
             stringstream ss(line);
             while(ss >> input)
             {
                 a[i][j] = input;
                 size_a++;
                 j++;
             }
             i++;
             rows_a++;
         }
         
         k=0;
         while(getline(file, line) && line != "")
         {
              j=0;
             stringstream ss(line);
             while(ss >> input)
             {
                 h[k][j] = input;
                 j++;
                 size_h++;
             }
             k++;
             rows_h++;
         }
     }
     file.close();
     columns_a = size_a/rows_a;
     columns_h = size_h/rows_h;
     int op_size = ((rows_a+rows_h-1)*(columns_a+columns_h-1));
     size_t size_ax = size_a*sizeof(double);
     size_t size_hx = size_h*sizeof(double);
     size_t size_cx = op_size*sizeof(double);
     
     // Allocate the host input vector a
     double *h_a = (double *)malloc(size_ax);
 
     // Allocate the host input vector h
     double *h_h = (double *)malloc(size_hx);
 
     // Allocate the host output vector c
     double *h_c = (double *)malloc(size_cx);

     // Verify that allocations succeeded
     if (h_a == NULL || h_h == NULL || h_c == NULL)
     {
         fprintf(stderr, "Failed to allocate host vectors!\n");
         exit(EXIT_FAILURE);
     }
     for (int i = 0; i < rows_a; i++)
     {
         for(int j=0;j< columns_a; j++)
         {
            h_a[i*columns_a + j] = a[i][j];
         }
     }

     for (int i = 0; i < rows_h; i++)
     {
         for(int j=0;j< columns_h; j++)
         {
            h_h[i*columns_h + j] = h[i][j];
         }
     }

     for (int i = 0; i < op_size; i++)
    {
        h_c[i] = rand()/(double)RAND_MAX;
    }
     //Allocate the device inputs
     double *d_a = NULL, *d_h = NULL, *d_c = NULL;
     err = hipMalloc((void **)&d_a, size_ax);
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate device vector a (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     err = hipMalloc((void **)&d_h, size_hx);
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate device vector h (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     err = hipMalloc((void **)&d_c, size_cx);
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate device vector c (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     // Copy the host input vectors a and h in host memory to the device input vectors in
     // device memory
     //printf("Copy input data from the host memory to the CUDA device\n");
     err=hipMemcpy(d_a, h_a, size_ax, hipMemcpyHostToDevice);
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to copy vector a from host to device (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     err = hipMemcpy(d_h, h_h, size_hx, hipMemcpyHostToDevice);
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     // Launch the CUDA Kernel
     dim3 blocksPerGrid(((rows_a + rows_h - 2) / 32) + 1, ((columns_h + columns_a - 2) / 32) + 1, 1);
     dim3 threadsPerBlock(32,32,1);
     convolution2D<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_h, d_c, rows_a, columns_a, rows_h, columns_h);
     err = hipGetLastError();
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to launch convolution2D kernel (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     err = hipDeviceSynchronize();
     if (err != hipSuccess)
	 {
		fprintf(stderr, "Failed to synchronize (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	 }
    /////////////////////////////////////////////////////////////////////////////////////////////////////////
     // Copy the device result in device memory to the host result in host memory.
     err = hipMemcpy(h_c, d_c, size_cx, hipMemcpyDeviceToHost);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to copy vector c from device to host (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     for (int i = 0; i < rows_a + rows_h - 1; i++) {
        for (int j = 0; j < columns_a + columns_h - 1; j++) 
        {	
			cout << fixed << setprecision(3) << h_c[(i*(columns_h+columns_a-1))+j] << " ";
		}
		cout << endl;
	}
 
     // Free device global memory
     err = hipFree(d_a);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to free device vector a (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     err = hipFree(d_h);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to free device vector h (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     err = hipFree(d_c);
 
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to free device vector c (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     // Free host memory
     free(h_a);
     free(h_h);
     free(h_c);
 
     // Reset the device and exit
     // cudaDeviceReset causes the driver to clean up all state. While
     // not mandatory in normal operation, it is good practice.  It is also
     // needed to ensure correct operation when the application is being
     // profiled. Calling cudaDeviceReset causes all profile data to be
     // flushed before the application exits
     err = hipDeviceReset();
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
 
     printf("Done\n");
     return 0;
 }
